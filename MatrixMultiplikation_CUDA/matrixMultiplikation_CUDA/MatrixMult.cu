#include "hip/hip_runtime.h"



#include <stdio.h>
#include <stdlib.h>
#include <iostream>

// 0,0 0,1 0,2
// 1,0 1,1 1,2
// => 0, 1, 2, 3, 4, 5
// => numberOfColumns * currentRow + currentColumn

__global__ void matrixmult(float* Cptr, float* Aptr, float* Bptr, int* m, int* n, int* upperbound, int *lowerbound) {
	for (int i = *lowerbound; i < *upperbound; i++) {
		for (int k = 0; k < *n; k++) {
			for (int j = 0; j < *n; j++) {
				Cptr[*n * i + k] += Aptr[*n * i + j] * Bptr[*n * j + k];
			}
		}
	}
}

float* createRandomMatrix(float *matrix, int m, int n) {
	matrix = new float[m * n];
	for (int r = 0; r < m; r++) {
		for (int c = 0; c < n; c++) {
			matrix[n * r + c] = static_cast <float> (rand() % 10);
		}
	}
	return matrix;
}

float* createEmptyMatrix(float* matrix, int m, int n) {
	matrix = new float[m * n];
	for (int r = 0; r < m; r++) {
		for (int c = 0; c < n; c++) {
			matrix[n * r + c] = 0;
		}
	}
	return matrix;
}

void print(float* matrix, int m, int n) {
	for (int r = 0; r < m; r++) {
		for (int c = 0; c < n; c++) {
			std::cout << matrix[n * r + c] << " ";
		}
		std::cout << "\n";
	}
	std::cout << "\n";
}

void deleteMatrix(float* matrix) {
	delete[] matrix;
}

int main() {

	int m;
	int n;
	int upperbound;
	int lowerbound;
	
	float* matrixA;
	float* matrixB;
	float* matrixC;

	float* d_matrixA;
	float* d_matrixB;
	float* d_matrixC;
	int* d_m;
	int* d_n;
	int* d_lowerbound;
	int* d_upperbound;

	//Allocate space for device copies
	hipMalloc((void**)&d_matrixA, (m * n) * sizeof(float));
	hipMalloc((void**)&d_matrixB, (m * n) * sizeof(float));
	hipMalloc((void**)&d_matrixC, (m * n) * sizeof(float));
	hipMalloc((void**)&d_m, sizeof(int));
	hipMalloc((void**)&d_n, sizeof(int));
	hipMalloc((void**)&d_lowerbound, sizeof(int));
	hipMalloc((void**)&d_upperbound, sizeof(int));

	m = 10;
	n = 10;
	lowerbound = 0;
	upperbound = m;
	matrixA = createRandomMatrix(matrixA, m, n);
	matrixB = createRandomMatrix(matrixB, m, n);
	matrixC = createEmptyMatrix(matrixC, m, n);

	/*print(matrixA, m, n);
	print(matrixB, m, n);*/

	hipMemcpy(d_matrixA, matrixA, (m * n) * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_matrixB, matrixB, (m * n) * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_matrixC, matrixC, (m * n) * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_m, &m, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_n, &n, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_lowerbound, &lowerbound, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_upperbound, &upperbound, sizeof(int), hipMemcpyHostToDevice);


	//Run Kernel on GPU
	matrixmult <<<1, 1 >>> (d_matrixC, d_matrixA, d_matrixB, d_m, d_n, d_upperbound, d_lowerbound);

	//Wait for GPU to finish
	hipDeviceSynchronize();

	/*matrixC = matrixmult(matrixC, matrixA, matrixB, m, n, m);*/
	hipMemcpy(matrixA, d_matrixA, (m * n) * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(matrixB, d_matrixB, (m * n) * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(matrixC, d_matrixC, (m * n) * sizeof(float), hipMemcpyDeviceToHost);
	/*cudaMemcpy(&m, d_m, sizeof(int), cudaMemcpyDeviceToHost);
	cudaMemcpy(&n, d_n, sizeof(int), cudaMemcpyDeviceToHost);*/



	print(matrixC, m, n); //ALWAYS 00000... WHY?!

	//Free memory
	hipFree(d_matrixA);
	hipFree(d_matrixB);
	hipFree(d_matrixC);
	hipFree(d_m);
	hipFree(d_n);
	
}