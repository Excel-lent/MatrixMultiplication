#include "hip/hip_runtime.h"



#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <chrono>

using namespace std::chrono;

// 0,0 0,1 0,2
// 1,0 1,1 1,2
// => 0, 1, 2, 3, 4, 5
// => numberOfColumns * currentRow + currentColumn

__global__ void matrixmult(float* Cptr, float* Aptr, float* Bptr, int m, int n) {
	// blockDim.x = number of threads in the current Block
	// threadIdx.x = index of current thread
	int Cidx = blockIdx.x * blockDim.x + threadIdx.x; // ^= n * i + k
	int i = Cidx / n;
	int k = Cidx - n * i;
	if (n * m > Cidx) {
		for (int j = 0; j < n; j++) {
			Cptr[Cidx] += Aptr[n * i + j] * Bptr[n * j + k];
		}
	}
}

float* createRandomMatrix(float *matrix, int m, int n) {
	matrix = new float[m * n];
	for (int r = 0; r < m; r++) {
		for (int c = 0; c < n; c++) {
			matrix[n * r + c] = static_cast <float> (rand() % 10) / 1.0;
		}
	}
	return matrix;
}

float* createEmptyMatrix(float* matrix, int m, int n) {
	matrix = new float[m * n];
	for (int r = 0; r < m; r++) {
		for (int c = 0; c < n; c++) {
			matrix[n * r + c] = 0.0;
		}
	}
	return matrix;
}

void print(float* matrix, int m, int n) {
	for (int r = 0; r < m; r++) {
		for (int c = 0; c < n; c++) {
			std::cout << matrix[n * r + c] << " ";
		}
		std::cout << "\n";
	}
	std::cout << "\n";
}

void deleteMatrix(float* matrix) {
	delete[] matrix;
}

int main() {

	int m = 1440;
	int n = 1440;
	int block_size = 512;
	
	//float pointer initialisieren und Speicher f�r den Array reservieren
	float* matrixA = (float*)malloc(m * n);
	float* matrixB = (float*)malloc(m * n);
	float* h_matrixC = (float*)malloc(m * n);

	float* d_matrixA;
	float* d_matrixB;
	float* d_matrixC;

	/*lowerbound = 0;
	upperbound = m;*/
	matrixA = createRandomMatrix(matrixA, m, n);
	matrixB = createRandomMatrix(matrixB, m, n);
	h_matrixC = createEmptyMatrix(h_matrixC, m, n);

	//Allocate space for device copies in device memory
	hipMalloc(&d_matrixA, (m * n) * sizeof(float));
	hipMalloc(&d_matrixB, (m * n) * sizeof(float));
	hipMalloc(&d_matrixC, (m * n) * sizeof(float));
	//cudaMalloc(&d_lowerbound, sizeof(int));
	//cudaMalloc(&d_upperbound, sizeof(int));

	//print(matrixA, m, n);
	//print(matrixB, m, n);

	hipMemcpy(d_matrixA, matrixA, (m * n) * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_matrixB, matrixB, (m * n) * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_matrixC, h_matrixC, (m * n) * sizeof(float), hipMemcpyHostToDevice);
	//cudaMemcpy(d_lowerbound, &lowerbound, sizeof(int), cudaMemcpyHostToDevice);
	//cudaMemcpy(d_upperbound, &upperbound, sizeof(int), cudaMemcpyHostToDevice);

	int Blocks = ((n*m) + block_size - 1) / block_size;
	std::cout << "[+] Calculation started with " << (Blocks * block_size) << " Threads";
	auto start = high_resolution_clock::now();
	//Run Kernel on GPU
	matrixmult <<<Blocks, block_size >>> (d_matrixC, d_matrixA, d_matrixB, m, n);

	//Wait for GPU to finish
	hipDeviceSynchronize();
	auto stop = high_resolution_clock::now();

	hipMemcpy(h_matrixC, d_matrixC, (m * n) * sizeof(float), hipMemcpyDeviceToHost);
	std::cout << "\n[+] Multithreaded calculation finished \n[+] Duration: " << duration<double>(stop - start).count() << " seconds";

	/*print(h_matrixC, m, n);*/

	//Free memory
	hipFree(d_matrixA);
	hipFree(d_matrixB);
	hipFree(d_matrixC);
	
	delete[] matrixA;
	delete[] matrixB;
	delete[] h_matrixC;
}